
#include <hip/hip_runtime.h>
#include <stdlib.h>

extern "C" __global__ void sum(const float* x, const float* y, float* out, int count) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
        out[i] = x[i] + y[i];
    }
}
